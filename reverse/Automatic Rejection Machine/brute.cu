
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define LENGTH(a) (sizeof(a) / sizeof(a[0]))

typedef struct hash_res {
        uint64_t a;
        uint64_t b;
#ifdef __cplusplus
        __device__ __host__ bool operator== (hash_res h) {
                return a = h.a && b == h.b;
        }
#endif
} HashRes;

__device__ HashRes somehash(int a, int b, int c) {
        HashRes h;
        int i;
        uint64_t r[4];
        uint64_t cv;


        h.a = a | (b << 8) | (c << 16) | 0xAABBCCDD11000000LL;
        h.b = 0x9E3779B912881288LL;

        r[0] = 0xDEADBEEFFEEDBEEFLL;
        r[1] = 0x1BADB002FACECAFELL;
        r[2] = 0xFEEDFACE08920892LL;
        r[3] = 0xCAFEFEED12401240LL;
        for (i = 0; i <= 0xf; ++i) {
                r[i & 3] = r[0] + r[1] + ((r[2] + r[3]) ^ (r[0] << (r[2] & 0x3f)));
                cv = r[i & 3];
                h.a += ((cv + h.b) << 9) ^ (cv - h.b) ^ ((cv + h.b) >> 14);
                h.b += ((cv + h.a) << 9) ^ (cv - h.a) ^ ((cv + h.a) >> 14);
        }
        return h;
}

__global__ void kernel(HashRes target, char *res) {
        uint64_t i;
        char a, b, c;
        const int index = threadIdx.x + (blockIdx.x<<10);
        const int stride = gridDim.x<<10;

        for (i = index; i < 0x1000000; i += stride) {
                a = (i >>  0) & 0xff;
                b = (i >>  8) & 0xff;
                c = (i >> 16) & 0xff;
                if (somehash(a, b, c) == target) {
                        res[0] = a;
                        res[1] = b;
                        res[2] = c;
                }
        }
}

void someshuffle(int *a) {
        int i;
        int j;
        int tmp;
        int dest[] = {0x0, 0x2, 0xb, 0x6, 0x4, 0x5, 0x3, 0x7, 0x8, 0x9, 0xa, 0x1, 0xc, 0x16, 0x18, 0xf, 0x11, 0x10, 0x12, 0x13, 0x17, 0x14, 0xd, 0x15, 0xe, 0x1d, 0x1c, 0x1b, 0x1a, 0x19};

        for (i = 0; i < 30; ++i) {
                for (j = i; dest[j] >= 0; j = tmp) {
                        tmp = a[i];
                        a[i] = a[dest[j]];
                        a[dest[j]] = tmp;
                        tmp = dest[j];
                        dest[j] = tmp - 30;
                }
        }
}

int main() {
        const HashRes targets[] = {
                {0xB4D8846071AC9EE5LL, 0x1E1FF00814E134FELL},
                {0x6B198E7941B7002ELL, 0xBC6FA839EFE36443LL},
                {0xC3C71AD9A664B6C3LL, 0x5692A2F09C98D986LL},
                {0xF084A1A59CD01E68LL, 0xBC52E78A7E4DF2DFLL},
                {0xDA219D93290B91A8LL, 0x5703D0286FA5D32FLL},
                {0x6274B1B118DA82B2LL, 0xA746EBFB0954EBBCLL},
                {0x5F6DF7BD4F1967A2LL, 0x16D5B5BDEE98CF8ELL},
                {0x52E8B6DF7E62E39ALL, 0x99F9455FB0C8D933LL},
                {0x5FFD82D53AF933DLL, 0xFF9084A16FF0141CLL},
                {0xE17C5F0781D52F9BLL, 0x1A0F4431548E51D1LL},
                {0xF2E8573D8F0F01DDLL, 0x250039177F4DEF91LL},
                {0x8851491ECBC7AF7CLL, 0xAD427C6695B91D24LL},
                {0x5E0071D97D98D094LL, 0x264DDA52B0C37B03LL},
                {0xA5811271D6D7C428LL, 0xE0133FC719F34136LL},
                {0xE508ACE2412B2633LL, 0x74321A3E9FACE34CLL},
                {0xFF5B8A59E8EBF70BLL, 0x76275A516F88C986LL},
                {0x1604D76F74599CC4LL, 0xF744BCD8F2016F58LL},
                {0xA0B6A7A0239E4EA7LL, 0xF1EFC57F15CB9AB4LL},
                {0xB0D1AD4FB4ED946ALL, 0x81CA31324D48E689LL},
                {0xE6A9979C51869F49LL, 0xA666637EE4BC2457LL},
                {0x6475B6AB4884B93CLL, 0x5C033B1207DA898FLL},
                {0xB66DC7E0DEC3443ELL, 0xE4899C99CFA0235CLL},
                {0x3B7FD8D4D0DCAF6BLL, 0xB1A4690DB34A7A7CLL},
                {0x8041D2607129ADABLL, 0xA6A1294A99894F1ALL},
                {0xDDE37A1C4524B831LL, 0x3BC8D81DE355B65CLL},
                {0x6C61AB15A63AD91ELL, 0x8FA4E37F4A3C7A39LL},
                {0x268B598404E773AFLL, 0x74F4F040AE13F867LL},
                {0x4DF78E91FD682404LL, 0xABE1FC425A9A671ALL},
                {0x1BB06615C8A31DD5LL, 0x9F56E9AEF2FA5D55LL},
                {0x239DCF030B3CE09BLL, 0x24556A34B61CA998LL},
        };
        int i;
        char *ans;
        char shuffled_flag[LENGTH(targets)] = {0};
        char flag[LENGTH(targets) + 1] = {0};
        int shuffle_order[LENGTH(targets) + 1] = {0};

        int device_id;
        int smcount;

        hipGetDevice(&device_id);
        hipDeviceGetAttribute(&smcount, hipDeviceAttributeMultiprocessorCount, device_id);


        hipMallocManaged(&ans, 3);
        for (i = 0; i < LENGTH(targets); i++) {
                kernel<<<smcount, 1024>>>(targets[i], ans);
                hipDeviceSynchronize();
                shuffled_flag[i] = ans[0];
        }
        hipFree(ans);


        for (i = 0; i < LENGTH(targets); i++) {
                shuffle_order[i] = i;
        }

        someshuffle(shuffle_order);

        for (i = 0; i < LENGTH(targets); i++) {
                flag[shuffle_order[i]] = shuffled_flag[i];
        }
        puts(flag);
}
